#include "hip/hip_runtime.h"
#ifndef DYNAMICS_CPP
#define DYNAMICS_CPP

#include "Dynamics.h"

KinematicsDynamics::KinematicsDynamics(BezierCurve* traj_input) {
    traj = traj_input;

    // pre-allocate memory
    links = PZsparseArray(NUM_FACTORS * 3, NUM_TIME_STEPS);
    mass_nominal_arr = PZsparseArray(NUM_JOINTS, 1);
    mass_uncertain_arr = PZsparseArray(NUM_JOINTS, 1);
    I_nominal_arr = PZsparseArray(NUM_JOINTS, 1);
    I_uncertain_arr = PZsparseArray(NUM_JOINTS, 1);
    u_nom = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);
    u_nom_int = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);

    // initialize robot properties
    for (int i = 0; i < NUM_JOINTS; i++) {
        trans_matrix(i, 0) = Eigen::MatrixXd::Zero(3, 1);
        trans_matrix(i, 0)(0) = trans[3 * i];
        trans_matrix(i, 0)(1) = trans[3 * i + 1];
        trans_matrix(i, 0)(2) = trans[3 * i + 2];

        // com_matrix(i, 0) = Eigen::MatrixXd::Zero(3, 1);
        // com_matrix(i, 0)(0) = com[3 * i];
        // com_matrix(i, 0)(1) = com[3 * i + 1];
        // com_matrix(i, 0)(2) = com[3 * i + 2];

        Eigen::MatrixXd mass_matrix(1, 1);
        mass_matrix(0) = mass[i];
        mass_nominal_arr(i) = PZsparse(mass_matrix);
        mass_uncertain_arr(i) = PZsparse(mass_matrix, mass_uncertainty);

        Eigen::Matrix3d inertia_matrix;
        for (int j = 0; j < 9; j++) {
            inertia_matrix(j) = inertia[i * 9 + j]; // This may not be right...
        }
        I_nominal_arr(i) = PZsparse(inertia_matrix);
        I_uncertain_arr(i) = PZsparse(inertia_matrix, inertia_uncertainty);
    }

    trans_matrix(NUM_JOINTS, 0) = Eigen::MatrixXd::Zero(3, 1);
    trans_matrix(NUM_JOINTS, 0)(0) = trans[3 * NUM_JOINTS];
    trans_matrix(NUM_JOINTS, 0)(1) = trans[3 * NUM_JOINTS + 1];
    trans_matrix(NUM_JOINTS, 0)(2) = trans[3 * NUM_JOINTS + 2];

    // define original link PZs
    links = PZsparseArray(NUM_JOINTS, NUM_TIME_STEPS);

    for (int i = 0; i < NUM_JOINTS; i++) {
        PZsparseArray link(3, 1);

        for (int j = 0; j < 3; j++) {
            uint64_t degree[1][NUM_FACTORS * 6] = {0};
            degree[0][NUM_FACTORS * (j + 1)] = 1; // use qde, qdae, qdde for x, y, z generator
            double temp = link_zonotope_generators[i][j];
            link(j, 0) = PZsparse(link_zonotope_center[i][j], &temp, degree, 1);
        }

        links(i, 0) = stack(link);

        for (int j = 1; j < NUM_TIME_STEPS; j++) {
            links(i, j) = links(i, 0);
        }
    }
}

void KinematicsDynamics::fk(uint s_ind) {
    PZsparse FK_R = PZsparse(0, 0, 0); // identity matrix
    PZsparse FK_T(3, 1);
    
    for (int i = 0; i < NUM_JOINTS; i++) {
        PZsparse P(trans_matrix(i, 0));
        
        FK_T = FK_T + FK_R * P;
        FK_R = FK_R * traj->R(i, s_ind);
        
        links(i, s_ind) = FK_R * links(i, s_ind) + FK_T;
    }
}

void KinematicsDynamics::rnea(uint s_ind,
                              PZsparseArray& mass_arr,
                              PZsparseArray& I_arr,
                              PZsparseArray& u,
                              bool setGravity) {
    PZsparse& cq1 = traj->cos_q_des(0, s_ind);
    PZsparse& cq2 = traj->cos_q_des(1, s_ind);
    PZsparse& cq3 = traj->cos_q_des(2, s_ind);
    PZsparse& cq4 = traj->cos_q_des(3, s_ind);
    PZsparse& cq5 = traj->cos_q_des(4, s_ind);
    PZsparse& cq6 = traj->cos_q_des(5, s_ind);
    PZsparse& cq7 = traj->cos_q_des(6, s_ind);

    PZsparse& sq1 = traj->sin_q_des(0, s_ind);
    PZsparse& sq2 = traj->sin_q_des(1, s_ind);
    PZsparse& sq3 = traj->sin_q_des(2, s_ind);
    PZsparse& sq4 = traj->sin_q_des(3, s_ind);
    PZsparse& sq5 = traj->sin_q_des(4, s_ind);
    PZsparse& sq6 = traj->sin_q_des(5, s_ind);
    PZsparse& sq7 = traj->sin_q_des(6, s_ind);

    PZsparse& qd1 = traj->qd_des(0, s_ind);
    PZsparse& qd2 = traj->qd_des(1, s_ind);
    PZsparse& qd3 = traj->qd_des(2, s_ind);
    PZsparse& qd4 = traj->qd_des(3, s_ind);
    PZsparse& qd5 = traj->qd_des(4, s_ind);
    PZsparse& qd6 = traj->qd_des(5, s_ind);
    PZsparse& qd7 = traj->qd_des(6, s_ind);

    PZsparse& qda1 = traj->qda_des(0, s_ind);
    PZsparse& qda2 = traj->qda_des(1, s_ind);
    PZsparse& qda3 = traj->qda_des(2, s_ind);
    PZsparse& qda4 = traj->qda_des(3, s_ind);
    PZsparse& qda5 = traj->qda_des(4, s_ind);
    PZsparse& qda6 = traj->qda_des(5, s_ind);
    PZsparse& qda7 = traj->qda_des(6, s_ind);

    PZsparse& qdd1 = traj->qdda_des(0, s_ind);
    PZsparse& qdd2 = traj->qdda_des(1, s_ind);
    PZsparse& qdd3 = traj->qdda_des(2, s_ind);
    PZsparse& qdd4 = traj->qdda_des(3, s_ind);
    PZsparse& qdd5 = traj->qdda_des(4, s_ind);
    PZsparse& qdd6 = traj->qdda_des(5, s_ind);
    PZsparse& qdd7 = traj->qdda_des(6, s_ind);

    PZsparse w1;
    PZsparse w2;
    PZsparse w3;
    PZsparse w_aux1;
    PZsparse w_aux2;
    PZsparse w_aux3;
    PZsparse wdot1;
    PZsparse wdot2;
    PZsparse wdot3;
    PZsparse linear_acc1;
    PZsparse linear_acc2;
    PZsparse linear_acc3;

    PZsparse w_new1;
    PZsparse w_new2;
    PZsparse w_new3;
    PZsparse w_aux_new1;
    PZsparse w_aux_new2;
    PZsparse w_aux_new3;
    PZsparse wdot_new1;
    PZsparse wdot_new2;
    PZsparse wdot_new3;
    PZsparse linear_acc_new1;
    PZsparse linear_acc_new2;
    PZsparse linear_acc_new3;

    PZsparse t1;
    PZsparse t2;
    PZsparse t3;
    PZsparse t4;
    PZsparse t5;
    PZsparse t6;
    PZsparse t7;
    PZsparse t8;
    PZsparse t9;
    PZsparse t10;
    PZsparse t11;
    PZsparse t12;
    PZsparse t13;
    PZsparse t14;
    PZsparse t15;
    PZsparse t16;
    PZsparse t17;

    // joint 1
    w_new3 = qd1;

    w_aux_new3 = qda1;

    wdot_new3 = qdd1;

    if (setGravity) {
        linear_acc_new3 = PZsparse(-gravity);
    }

    w3 = w_new3;
    w_aux3 = w_aux_new3;
    wdot3 = wdot_new3;
    linear_acc3 = linear_acc_new3;

    t2 = com[0][0]*w_aux2;
    t3 = com[0][0]*w_aux3;
    t4 = com[0][1]*w_aux1;
    t5 = com[0][1]*w_aux3;
    t6 = com[0][2]*w_aux1;
    t7 = com[0][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F1_1 = -mass_arr(0, 0)*(-linear_acc1+com[0][1]*wdot3-com[0][2]*wdot2+t11*w2+t12*w3);
    PZsparse F1_2 = mass_arr(0, 0)*(linear_acc2+com[0][0]*wdot3-com[0][2]*wdot1+t11*w1-t13*w3);
    PZsparse F1_3 = mass_arr(0, 0)*(linear_acc3-com[0][0]*wdot2+com[0][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(0,0)*w1;
    t3 = I_arr(0,1)*w2;
    t4 = I_arr(0,2)*w3;
    t5 = I_arr(0,3)*w1;
    t6 = I_arr(0,4)*w2;
    t7 = I_arr(0,5)*w3;
    t8 = I_arr(0,6)*w1;
    t9 = I_arr(0,7)*w2;
    t10 = I_arr(0,8)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N1_1 = I_arr(0,0)*wdot1+I_arr(0,1)*wdot2+I_arr(0,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N1_2 = I_arr(0,3)*wdot1+I_arr(0,4)*wdot2+I_arr(0,5)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N1_3 = I_arr(0,6)*wdot1+I_arr(0,7)*wdot2+I_arr(0,8)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 2
    w_new1 = cq2*w1+sq2*w3;
    w_new2 = cq2*w3-sq2*w1;
    w_new3 = qd2-w2;

    w_aux_new1 = cq2*w_aux1+sq2*w_aux3;
    w_aux_new2 = cq2*w_aux3-sq2*w_aux1;
    w_aux_new3 = qda2-w_aux2;

    wdot_new1 = cq2*wdot1+sq2*wdot3+qd2*(cq2*w_aux3-sq2*w_aux1);
    wdot_new2 = cq2*wdot3-sq2*wdot1-qd2*(cq2*w_aux1+sq2*w_aux3);
    wdot_new3 = qdd2-wdot2;

    t2 = -linear_acc1;
    t3 = w_aux3*5.375E-3;
    t4 = wdot1*5.375E-3;
    t5 = wdot3*5.375E-3;
    t6 = w_aux1*w2*5.375E-3;
    t8 = w_aux2*1.2838E-1;
    t9 = wdot2*1.2838E-1;
    t10 = w_aux1*w1*1.2838E-1;
    t11 = w_aux1*w3*1.2838E-1;
    t7 = -t6;
    t12 = t3+t8;
    t13 = t12*w2;
    t15 = t2+t5+t7+t9+t11;
    t14 = linear_acc3+t4+t10+t13;
    linear_acc_new1 = -cq2*t15+sq2*t14;
    linear_acc_new2 = cq2*t14+sq2*t15;
    linear_acc_new3 = -linear_acc2-wdot1*1.2838E-1+t12*w3+w_aux1*w1*5.375E-3;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[1][0]*w_aux2;
    t3 = com[1][0]*w_aux3;
    t4 = com[1][1]*w_aux1;
    t5 = com[1][1]*w_aux3;
    t6 = com[1][2]*w_aux1;
    t7 = com[1][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F2_1 = -mass_arr(1, 0)*(-linear_acc1+com[1][1]*wdot3-com[1][2]*wdot2+t11*w2+t12*w3);
    PZsparse F2_2 = mass_arr(1, 0)*(linear_acc2+com[1][0]*wdot3-com[1][2]*wdot1+t11*w1-t13*w3);
    PZsparse F2_3 = mass_arr(1, 0)*(linear_acc3-com[1][0]*wdot2+com[1][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(1,0)*w1;
    t3 = I_arr(1,1)*w2;
    t4 = I_arr(1,2)*w3;
    t5 = I_arr(1,3)*w1;
    t6 = I_arr(1,4)*w2;
    t7 = I_arr(1,5)*w3;
    t8 = I_arr(1,6)*w1;
    t9 = I_arr(1,7)*w2;
    t10 = I_arr(1,8)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N2_1 = I_arr(1,0)*wdot1+I_arr(1,1)*wdot2+I_arr(1,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N2_2 = I_arr(1,3)*wdot1+I_arr(1,4)*wdot2+I_arr(1,5)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N2_3 = I_arr(1,6)*wdot1+I_arr(1,7)*wdot2+I_arr(1,8)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 3
    w_new1 = cq3*w1-sq3*w3;
    w_new2 = -cq3*w3-sq3*w1;
    w_new3 = qd3+w2;

    w_aux_new1 = cq3*w_aux1-sq3*w_aux3;
    w_aux_new2 = -cq3*w_aux3-sq3*w_aux1;
    w_aux_new3 = qda3+w_aux2;

    wdot_new1 = cq3*wdot1-sq3*wdot3-qd3*(cq3*w_aux3+sq3*w_aux1);
    wdot_new2 = -cq3*wdot3-sq3*wdot1-qd3*(cq3*w_aux1-sq3*w_aux3);
    wdot_new3 = qdd3+wdot2;

    t2 = -linear_acc1;
    t3 = w_aux2*6.375E-3;
    t4 = wdot2*6.375E-3;
    t5 = w_aux1*w1*6.375E-3;
    t6 = w_aux1*w3*6.375E-3;
    t7 = w_aux3*2.1038E-1;
    t8 = wdot1*2.1038E-1;
    t9 = wdot3*2.1038E-1;
    t10 = w_aux1*w2*2.1038E-1;
    t11 = -t7;
    t12 = -t8;
    t13 = -t9;
    t14 = t3+t11;
    t16 = t2+t4+t6+t10+t13;
    t15 = t14*w2;
    t17 = linear_acc3+t5+t12+t15;
    linear_acc_new1 = -cq3*t16-sq3*t17;
    linear_acc_new2 = -cq3*t17+sq3*t16;
    linear_acc_new3 = linear_acc2+wdot1*6.375E-3-t14*w3+w_aux1*w1*2.1038E-1;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[2][0]*w_aux2;
    t3 = com[2][0]*w_aux3;
    t4 = com[2][1]*w_aux1;
    t5 = com[2][1]*w_aux3;
    t6 = com[2][2]*w_aux1;
    t7 = com[2][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F3_1 = -mass_arr(2, 0)*(-linear_acc1+com[2][1]*wdot3-com[2][2]*wdot2+t11*w2+t12*w3);
    PZsparse F3_2 = mass_arr(2, 0)*(linear_acc2+com[2][0]*wdot3-com[2][2]*wdot1+t11*w1-t13*w3);
    PZsparse F3_3 = mass_arr(2, 0)*(linear_acc3-com[2][0]*wdot2+com[2][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(2,0)*w1;
    t3 = I_arr(2,1)*w2;
    t4 = I_arr(2,2)*w3;
    t5 = I_arr(2,3)*w1;
    t6 = I_arr(2,4)*w2;
    t7 = I_arr(2,5)*w3;
    t8 = I_arr(2,6)*w1;
    t9 = I_arr(2,7)*w2;
    t10 = I_arr(2,8)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N3_1 = I_arr(2,0)*wdot1+I_arr(2,1)*wdot2+I_arr(2,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N3_2 = I_arr(2,3)*wdot1+I_arr(2,4)*wdot2+I_arr(2,5)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N3_3 = I_arr(2,6)*wdot1+I_arr(2,7)*wdot2+I_arr(2,8)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 4
    w_new1 = cq4*w1+sq4*w3;
    w_new2 = cq4*w3-sq4*w1;
    w_new3 = qd4-w2;

    w_aux_new1 = cq4*w_aux1+sq4*w_aux3;
    w_aux_new2 = cq4*w_aux3-sq4*w_aux1;
    w_aux_new3 = qda4-w_aux2;

    wdot_new1 = cq4*wdot1+sq4*wdot3+qd4*(cq4*w_aux3-sq4*w_aux1);
    wdot_new2 = cq4*wdot3-sq4*wdot1-qd4*(cq4*w_aux1+sq4*w_aux3);
    wdot_new3 = qdd4-wdot2;

    t2 = -linear_acc1;
    t3 = w_aux3*6.375E-3;
    t4 = wdot1*6.375E-3;
    t5 = wdot3*6.375E-3;
    t6 = w_aux1*w2*6.375E-3;
    t8 = w_aux2*2.1038E-1;
    t9 = wdot2*2.1038E-1;
    t10 = w_aux1*w1*2.1038E-1;
    t11 = w_aux1*w3*2.1038E-1;
    t7 = -t6;
    t12 = t3+t8;
    t13 = t12*w2;
    t15 = t2+t5+t7+t9+t11;
    t14 = linear_acc3+t4+t10+t13;
    linear_acc_new1 = -cq4*t15+sq4*t14;
    linear_acc_new2 = cq4*t14+sq4*t15;
    linear_acc_new3 = -linear_acc2-wdot1*2.1038E-1+t12*w3+w_aux1*w1*6.375E-3;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[3][0]*w_aux2;
    t3 = com[3][0]*w_aux3;
    t4 = com[3][1]*w_aux1;
    t5 = com[3][1]*w_aux3;
    t6 = com[3][2]*w_aux1;
    t7 = com[3][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F4_1 = -mass_arr(3, 0)*(-linear_acc1+com[3][1]*wdot3-com[3][2]*wdot2+t11*w2+t12*w3);
    PZsparse F4_2 = mass_arr(3, 0)*(linear_acc2+com[3][0]*wdot3-com[3][2]*wdot1+t11*w1-t13*w3);
    PZsparse F4_3 = mass_arr(3, 0)*(linear_acc3-com[3][0]*wdot2+com[3][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(3,0)*w1;
    t3 = I_arr(3,1)*w2;
    t4 = I_arr(3,2)*w3;
    t5 = I_arr(3,3)*w1;
    t6 = I_arr(3,4)*w2;
    t7 = I_arr(3,5)*w3;
    t8 = I_arr(3,6)*w1;
    t9 = I_arr(3,7)*w2;
    t10 = I_arr(3,8)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N4_1 = I_arr(3,0)*wdot1+I_arr(3,1)*wdot2+I_arr(3,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N4_2 = I_arr(3,3)*wdot1+I_arr(3,4)*wdot2+I_arr(3,5)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N4_3 = I_arr(3,6)*wdot1+I_arr(3,7)*wdot2+I_arr(3,8)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 5
    w_new1 = cq5*w1-sq5*w3;
    w_new2 = -cq5*w3-sq5*w1;
    w_new3 = qd5+w2;

    w_aux_new1 = cq5*w_aux1-sq5*w_aux3;
    w_aux_new2 = -cq5*w_aux3-sq5*w_aux1;
    w_aux_new3 = qda5+w_aux2;

    wdot_new1 = cq5*wdot1-sq5*wdot3-qd5*(cq5*w_aux3+sq5*w_aux1);
    wdot_new2 = -cq5*wdot3-sq5*wdot1-qd5*(cq5*w_aux1-sq5*w_aux3);
    wdot_new3 = qdd5+wdot2;

    t2 = -linear_acc1;
    t3 = w_aux2*6.375E-3;
    t4 = wdot2*6.375E-3;
    t5 = w_aux1*w1*6.375E-3;
    t6 = w_aux1*w3*6.375E-3;
    t7 = w_aux3*2.0843E-1;
    t8 = wdot1*2.0843E-1;
    t9 = wdot3*2.0843E-1;
    t10 = w_aux1*w2*2.0843E-1;
    t11 = -t7;
    t12 = -t8;
    t13 = -t9;
    t14 = t3+t11;
    t16 = t2+t4+t6+t10+t13;
    t15 = t14*w2;
    t17 = linear_acc3+t5+t12+t15;
    linear_acc_new1 = -cq5*t16-sq5*t17;
    linear_acc_new2 = -cq5*t17+sq5*t16;
    linear_acc_new3 = linear_acc2+wdot1*6.375E-3-t14*w3+w_aux1*w1*2.0843E-1;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[4][0]*w_aux2;
    t3 = com[4][0]*w_aux3;
    t4 = com[4][1]*w_aux1;
    t5 = com[4][1]*w_aux3;
    t6 = com[4][2]*w_aux1;
    t7 = com[4][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F5_1 = -mass_arr(4, 0)*(-linear_acc1+com[4][1]*wdot3-com[4][2]*wdot2+t11*w2+t12*w3);
    PZsparse F5_2 = mass_arr(4, 0)*(linear_acc2+com[4][0]*wdot3-com[4][2]*wdot1+t11*w1-t13*w3);
    PZsparse F5_3 = mass_arr(4, 0)*(linear_acc3-com[4][0]*wdot2+com[4][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(4,0)*w1;
    t3 = I_arr(4,1)*w2;
    t4 = I_arr(4,2)*w3;
    t5 = I_arr(4,3)*w1;
    t6 = I_arr(4,4)*w2;
    t7 = I_arr(4,5)*w3;
    t8 = I_arr(4,6)*w1;
    t9 = I_arr(4,7)*w2;
    t10 = I_arr(4,8)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N5_1 = I_arr(4,0)*wdot1+I_arr(4,1)*wdot2+I_arr(4,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N5_2 = I_arr(4,3)*wdot1+I_arr(4,4)*wdot2+I_arr(4,5)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N5_3 = I_arr(4,6)*wdot1+I_arr(4,7)*wdot2+I_arr(4,8)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 6
    w_new1 = cq6*w1+sq6*w3;
    w_new2 = cq6*w3-sq6*w1;
    w_new3 = qd6-w2;

    w_aux_new1 = cq6*w_aux1+sq6*w_aux3;
    w_aux_new2 = cq6*w_aux3-sq6*w_aux1;
    w_aux_new3 = qda6-w_aux2;

    wdot_new1 = cq6*wdot1+sq6*wdot3+qd6*(cq6*w_aux3-sq6*w_aux1);
    wdot_new2 = cq6*wdot3-sq6*wdot1-qd6*(cq6*w_aux1+sq6*w_aux3);
    wdot_new3 = qdd6-wdot2;

    t2 = -linear_acc1;
    t3 = w_aux2*1.0593E-1;
    t4 = wdot2*1.0593E-1;
    t5 = w_aux1*w1*1.0593E-1;
    t6 = w_aux1*w3*1.0593E-1;
    t7 = w_aux3*1.750499999999995E-4;
    t8 = wdot1*1.750499999999995E-4;
    t9 = wdot3*1.750499999999995E-4;
    t10 = w_aux1*w2*1.750499999999995E-4;
    t11 = -t10;
    t12 = t3+t7;
    t13 = t12*w2;
    t15 = t2+t4+t6+t9+t11;
    t14 = linear_acc3+t5+t8+t13;
    linear_acc_new1 = -cq6*t15+sq6*t14;
    linear_acc_new2 = cq6*t14+sq6*t15;
    linear_acc_new3 = -linear_acc2-wdot1*1.0593E-1+t12*w3+w_aux1*w1*1.750499999999995E-4;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[5][0]*w_aux2;
    t3 = com[5][0]*w_aux3;
    t4 = com[5][1]*w_aux1;
    t5 = com[5][1]*w_aux3;
    t6 = com[5][2]*w_aux1;
    t7 = com[5][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F6_1 = -mass_arr(5, 0)*(-linear_acc1+com[5][1]*wdot3-com[5][2]*wdot2+t11*w2+t12*w3);
    PZsparse F6_2 = mass_arr(5, 0)*(linear_acc2+com[5][0]*wdot3-com[5][2]*wdot1+t11*w1-t13*w3);
    PZsparse F6_3 = mass_arr(5, 0)*(linear_acc3-com[5][0]*wdot2+com[5][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(5,0)*w1;
    t3 = I_arr(5,1)*w2;
    t4 = I_arr(5,2)*w3;
    t5 = I_arr(5,3)*w1;
    t6 = I_arr(5,4)*w2;
    t7 = I_arr(5,5)*w3;
    t8 = I_arr(5,6)*w1;
    t9 = I_arr(5,7)*w2;
    t10 = I_arr(5,8)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N6_1 = I_arr(5,0)*wdot1+I_arr(5,1)*wdot2+I_arr(5,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N6_2 = I_arr(5,3)*wdot1+I_arr(5,4)*wdot2+I_arr(5,5)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N6_3 = I_arr(5,6)*wdot1+I_arr(5,7)*wdot2+I_arr(5,8)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 7
    w_new1 = cq7*w1-sq7*w3;
    w_new2 = -cq7*w3-sq7*w1;
    w_new3 = qd7+w2;

    w_aux_new1 = cq7*w_aux1-sq7*w_aux3;
    w_aux_new2 = -cq7*w_aux3-sq7*w_aux1;
    w_aux_new3 = qda7+w_aux2;

    wdot_new1 = cq7*wdot1-sq7*wdot3-qd7*(cq7*w_aux3+sq7*w_aux1);
    wdot_new2 = -cq7*wdot3-sq7*wdot1-qd7*(cq7*w_aux1-sq7*w_aux3);
    wdot_new3 = qdd7+wdot2;

    t2 = -linear_acc1;
    t3 = w_aux3*1.0593E-1;
    t4 = wdot1*1.0593E-1;
    t5 = wdot3*1.0593E-1;
    t6 = w_aux1*w2*1.0593E-1;
    t10 = w_aux2*1.75050000000003E-4;
    t11 = wdot2*1.75050000000003E-4;
    t12 = w_aux1*w1*1.75050000000003E-4;
    t13 = w_aux1*w3*1.75050000000003E-4;
    t7 = -t3;
    t8 = -t4;
    t9 = -t5;
    t15 = -w2*(t3-t10);
    t14 = t7+t10;
    t16 = linear_acc3+t8+t12+t15;
    t17 = t2+t6+t9+t11+t13;
    linear_acc_new1 = -cq7*t17-sq7*t16;
    linear_acc_new2 = -cq7*t16+sq7*t17;
    linear_acc_new3 = linear_acc2+wdot1*1.75050000000003E-4+w_aux1*w1*1.0593E-1+w3*(t3-t10);

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[6][0]*w_aux2;
    t3 = com[6][0]*w_aux3;
    t4 = com[6][1]*w_aux1;
    t5 = com[6][1]*w_aux3;
    t6 = com[6][2]*w_aux1;
    t7 = com[6][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F7_1 = -mass_arr(6, 0)*(-linear_acc1+com[6][1]*wdot3-com[6][2]*wdot2+t11*w2+t12*w3);
    PZsparse F7_2 = mass_arr(6, 0)*(linear_acc2+com[6][0]*wdot3-com[6][2]*wdot1+t11*w1-t13*w3);
    PZsparse F7_3 = mass_arr(6, 0)*(linear_acc3-com[6][0]*wdot2+com[6][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(6,0)*w1;
    t3 = I_arr(6,1)*w2;
    t4 = I_arr(6,2)*w3;
    t5 = I_arr(6,3)*w1;
    t6 = I_arr(6,4)*w2;
    t7 = I_arr(6,5)*w3;
    t8 = I_arr(6,6)*w1;
    t9 = I_arr(6,7)*w2;
    t10 = I_arr(6,8)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N7_1 = I_arr(6,0)*wdot1+I_arr(6,1)*wdot2+I_arr(6,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N7_2 = I_arr(6,3)*wdot1+I_arr(6,4)*wdot2+I_arr(6,5)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N7_3 = I_arr(6,6)*wdot1+I_arr(6,7)*wdot2+I_arr(6,8)*wdot3-t11*w_aux2+t12*w_aux1;

    PZsparse f7_1 = F7_1;
    PZsparse f7_2 = F7_2;
    PZsparse f7_3 = F7_3;

    PZsparse n7_1 = N7_1+F7_3*com[6][1]-F7_2*com[6][2];
    PZsparse n7_2 = N7_2-F7_3*com[6][0]+F7_1*com[6][2];
    PZsparse n7_3 = N7_3+F7_2*com[6][0]-F7_1*com[6][1];

    PZsparse f6_1 = F6_1+cq7*f7_1-f7_2*sq7;
    PZsparse f6_2 = F6_2+f7_3;
    PZsparse f6_3 = F6_3-cq7*f7_2-f7_1*sq7;

    PZsparse  n6_1 = N6_1+f7_3*1.75050000000003E-4+F6_3*com[5][1]-F6_2*com[5][2]+cq7*f7_2*1.0593E-1+cq7*n7_1+f7_1*sq7*1.0593E-1-n7_2*sq7;
    PZsparse  n6_2 = N6_2+n7_3-F6_3*com[5][0]+F6_1*com[5][2]-cq7*f7_1*1.75050000000003E-4+f7_2*sq7*1.75050000000003E-4;
    PZsparse  n6_3 = N6_3+F6_2*com[5][0]-F6_1*com[5][1]+cq7*f7_1*1.0593E-1-cq7*n7_2-f7_2*sq7*1.0593E-1-n7_1*sq7;

    PZsparse f5_1 = F5_1+cq6*f6_1-f6_2*sq6;
    PZsparse f5_2 = F5_2-f6_3;
    PZsparse f5_3 = F5_3+cq6*f6_2+f6_1*sq6;

    PZsparse  n5_1 = N5_1-f6_3*1.0593E-1+F5_3*com[4][1]-F5_2*com[4][2]+cq6*f6_2*1.750499999999995E-4+cq6*n6_1+f6_1*sq6*1.750499999999995E-4-n6_2*sq6;
    PZsparse  n5_2 = N5_2-n6_3-F5_3*com[4][0]+F5_1*com[4][2]-cq6*f6_1*1.0593E-1+f6_2*sq6*1.0593E-1;
    PZsparse  n5_3 = N5_3+F5_2*com[4][0]-F5_1*com[4][1]-cq6*f6_1*1.750499999999995E-4+cq6*n6_2+f6_2*sq6*1.750499999999995E-4+n6_1*sq6;

    PZsparse f4_1 = F4_1+cq5*f5_1-f5_2*sq5;
    PZsparse f4_2 = F4_2+f5_3;
    PZsparse f4_3 = F4_3-cq5*f5_2-f5_1*sq5;

    PZsparse  n4_1 = N4_1+f5_3*6.375E-3+F4_3*com[3][1]-F4_2*com[3][2]+cq5*f5_2*2.0843E-1+cq5*n5_1+f5_1*sq5*2.0843E-1-n5_2*sq5;
    PZsparse  n4_2 = N4_2+n5_3-F4_3*com[3][0]+F4_1*com[3][2]-cq5*f5_1*6.375E-3+f5_2*sq5*6.375E-3;
    PZsparse  n4_3 = N4_3+F4_2*com[3][0]-F4_1*com[3][1]+cq5*f5_1*2.0843E-1-cq5*n5_2-f5_2*sq5*2.0843E-1-n5_1*sq5;

    PZsparse f3_1 = F3_1+cq4*f4_1-f4_2*sq4;
    PZsparse f3_2 = F3_2-f4_3;
    PZsparse f3_3 = F3_3+cq4*f4_2+f4_1*sq4;

    PZsparse  n3_1 = N3_1-f4_3*2.1038E-1+F3_3*com[2][1]-F3_2*com[2][2]+cq4*f4_2*6.375E-3+cq4*n4_1+f4_1*sq4*6.375E-3-n4_2*sq4;
    PZsparse  n3_2 = N3_2-n4_3-F3_3*com[2][0]+F3_1*com[2][2]-cq4*f4_1*2.1038E-1+f4_2*sq4*2.1038E-1;
    PZsparse  n3_3 = N3_3+F3_2*com[2][0]-F3_1*com[2][1]-cq4*f4_1*6.375E-3+cq4*n4_2+f4_2*sq4*6.375E-3+n4_1*sq4;

    PZsparse f2_1 = F2_1+cq3*f3_1-f3_2*sq3;
    PZsparse f2_2 = F2_2+f3_3;
    PZsparse f2_3 = F2_3-cq3*f3_2-f3_1*sq3;

    PZsparse  n2_1 = N2_1+f3_3*6.375E-3+F2_3*com[1][1]-F2_2*com[1][2]+cq3*f3_2*2.1038E-1+cq3*n3_1+f3_1*sq3*2.1038E-1-n3_2*sq3;
    PZsparse  n2_2 = N2_2+n3_3-F2_3*com[1][0]+F2_1*com[1][2]-cq3*f3_1*6.375E-3+f3_2*sq3*6.375E-3;
    PZsparse  n2_3 = N2_3+F2_2*com[1][0]-F2_1*com[1][1]+cq3*f3_1*2.1038E-1-cq3*n3_2-f3_2*sq3*2.1038E-1-n3_1*sq3;

    PZsparse f1_1 = F1_1+cq2*f2_1-f2_2*sq2;
    PZsparse f1_2 = F1_2-f2_3;
    PZsparse f1_3 = F1_3+cq2*f2_2+f2_1*sq2;

    PZsparse n1_1 = N1_1-f2_3*1.2838E-1+F1_3*com[0][1]-F1_2*com[0][2]+cq2*f2_2*5.375E-3+cq2*n2_1+f2_1*sq2*5.375E-3-n2_2*sq2;
    PZsparse n1_2 = N1_2-n2_3-F1_3*com[0][0]+F1_1*com[0][2]-cq2*f2_1*1.2838E-1+f2_2*sq2*1.2838E-1;
    PZsparse n1_3 = N1_3+F1_2*com[0][0]-F1_1*com[0][1]-cq2*f2_1*5.375E-3+cq2*n2_2+f2_2*sq2*5.375E-3+n2_1*sq2;

    u(1, 0) = n1_3;
    u(2, 0) = n2_3;
    u(3, 0) = n3_3;
    u(4, 0) = n4_3;
    u(5, 0) = n5_3;
    u(6, 0) = n6_3;
    u(7, 0) = n7_3;
}

// void KinematicsDynamics::rnea(uint s_ind,
//                               PZsparseArray& mass_arr,
//                               PZsparseArray& I_arr,
//                               PZsparseArray& u,
//                               bool setGravity) {
//     PZsparse w(3, 1);
//     PZsparse wdot(3, 1);
//     PZsparse w_aux(3, 1);
//     PZsparse linear_acc(3, 1);

//     PZsparseArray F(NUM_JOINTS, 1);
//     PZsparseArray N(NUM_JOINTS, 1);

//     if (setGravity) { // set gravity
//         // directly modify the center of the PZ instance
//         linear_acc.center(2) = gravity;
//     }

//     // RNEA forward recursion
//     for (int i = 0; i < NUM_JOINTS; i++) {
//         // NOTE:
//         // This is just a simplified implementation!!!
//         // We assume all fixed joints are at the end and the revolute joints are consecutive
//         if (axes[i] != 0) { // revolute joints
//             // line 16
//             linear_acc = traj->R_t(i, s_ind) * (linear_acc 
//                                                  + cross(wdot, trans_matrix(i, 0)) 
//                                                  + cross(w, cross(w_aux, trans_matrix(i, 0))));

//             // line 13
//             w = traj->R_t(i, s_ind) * w;
//             w.addOneDimPZ(traj->qd_des(i, s_ind), abs(axes[i]) - 1, 0);

//             // line 14
//             w_aux = traj->R_t(i, s_ind) * w_aux;

//             // line 15
//             wdot = traj->R_t(i, s_ind) * wdot;

//             PZsparse temp(3, 1); // temp = joint_vel(robot_params.q_index(i))*z(:,i)
//             temp.addOneDimPZ(traj->qd_des(i, s_ind), abs(axes[i]) - 1, 0);

//             wdot = wdot + cross(w_aux, temp);

//             wdot.addOneDimPZ(traj->qdda_des(i, s_ind), abs(axes[i]) - 1, 0);

//             // line 14
//             w_aux.addOneDimPZ(traj->qda_des(i, s_ind), abs(axes[i]) - 1, 0);
//         }
//         else { // fixed joints
//             // line 16
//             linear_acc = traj->R_t(i, s_ind) * (linear_acc 
//                                                  + cross(wdot, trans_matrix(i, 0)) 
//                                                  + cross(w, cross(w_aux, trans_matrix(i, 0))));

//             // line 13
//             w = traj->R_t(i, s_ind) * w;

//             // line 14
//             w_aux = traj->R_t(i, s_ind) * w_aux;

//             // line 15
//             wdot = traj->R_t(i, s_ind) * wdot;
//         }

//         // line 23 & 27
//         F(i, 0) = mass_arr(i, 0) * (linear_acc
//                                      + cross(wdot, com_matrix(i, 0))
//                                      + cross(w, cross(w_aux, com_matrix(i, 0))));

//         // line 29
//         N(i, 0) = I_arr(i, 0) * wdot + cross(w_aux, (I_arr(i, 0) * w));
//     }

//     PZsparse f(3, 1);
//     PZsparse n(3, 1);

//     // RNEA reverse recursion
//     for (int i = NUM_JOINTS - 1; i >= 0; i--) {
//         // line 29
//         n = N(i, 0)
//             + traj->R(i + 1, s_ind) * n
//             + cross(com_matrix(i, 0), F(i, 0))
//             + cross(trans_matrix(i + 1, 0), traj->R(i + 1, s_ind) * f);

//         // line 28
//         f = traj->R(i + 1, s_ind) * f + F(i, 0);

//         if (axes[i] != 0) {
//             u(i, s_ind) = n(abs(axes[i]) - 1, 0);

//             u(i, s_ind) = u(i, s_ind) + armature[i] * traj->qdda_des(i, s_ind);

//             u(i, s_ind) = u(i, s_ind) + damping[i] * traj->qd_des(i, s_ind);

//             // friction is directly cut on the torque limits
//         }
//     }
// }

#endif